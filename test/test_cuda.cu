#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016, Dalian Futures Information Technology Co., Ltd.
 *
 * Bo Wang
 * Xiaoye Meng <mengxiaoye at dce dot com dot cn>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
	int count = 0, dev;

	hipGetDeviceCount(&count);
	if (count == 0)
		fprintf(stdout, "There are no available device(s) that support CUDA\n");
	else
		fprintf(stdout, "Detected %d CUDA Capable device(s)\n", count);
	for (dev = 0; dev < count; ++dev) {
		hipDeviceProp_t prop;
		int dver = 0, rver = 0;

		hipSetDevice(dev);
		hipGetDeviceProperties(&prop, dev);
		fprintf(stdout, "\nDevice %d: \"%s\"\n", dev, prop.name);
		hipDriverGetVersion(&dver);
		hipRuntimeGetVersion(&rver);
		fprintf(stdout, "CUDA Driver Version / Runtime Version"
			"                                               %d.%d / %d.%d\n",
			dver / 1000, dver % 100 / 10, rver / 1000, rver % 100 / 10);
		fprintf(stdout, "CUDA Major / Minor compute capability:"
			"                                              %d.%d\n", prop.major, prop.minor);
		fprintf(stdout, "Device has ECC support enabled:"
			"                                                     %d\n", prop.ECCEnabled);
		fprintf(stdout, "Number of asynchronous engines:"
			"                                                     %d\n", prop.asyncEngineCount);
		fprintf(stdout, "Device can map host memory with hipHostAlloc/hipHostGetDevicePointer:"
			"             %d\n", prop.canMapHostMemory);
		fprintf(stdout, "Clock frequency in kilohertz:"
			"                                                       %d\n", prop.clockRate);
		fprintf(stdout, "Compute mode:           "
			"                                                            %d\n", prop.computeMode);
		fprintf(stdout, "Device can possibly execute multiple kernels concurrently:"
			"                          %d\n", prop.concurrentKernels);
		fprintf(stdout, "Device supports caching globals in L1:"
			"                                              %d\n", prop.globalL1CacheSupported);
		fprintf(stdout, "Device is integrated as opposed to discrete:"
			"                                        %d\n", prop.integrated);
		fprintf(stdout, "Device is on a multi-GPU board:"
			"                                                     %d\n", prop.isMultiGpuBoard);
		fprintf(stdout, "Specified whether there is a run time limit on kernels:"
			"                             %d\n", prop.kernelExecTimeoutEnabled);
		fprintf(stdout, "Size of L2 cache in bytes:"
			"                                                          %d\n", prop.l2CacheSize);
		fprintf(stdout, "Device supports caching locals in L1:"
			"                                               %d\n", prop.localL1CacheSupported);
		fprintf(stdout, "Device supports allocating managed memory on this system:"
			"                           %d\n", prop.managedMemory);
		fprintf(stdout, "Maximum size of each dimension of a grid:"
			"                                           (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		fprintf(stdout, "Maximum 1D surface size: "
			"                                                           %d\n", prop.maxSurface1D);
		fprintf(stdout, "Maximum 1D layered surface dimensions:"
			"                                              (%d, %d)\n",
			prop.maxSurface1DLayered[0], prop.maxSurface1DLayered[1]);
		fprintf(stdout, "Maximum 2D surface dimensions:"
			"                                                      (%d, %d)\n",
			prop.maxSurface2D[0], prop.maxSurface2D[1]);
		fprintf(stdout, "Maximum 2D layered surface dimensions:"
			"                                              (%d, %d, %d)\n",
			prop.maxSurface2DLayered[0], prop.maxSurface2DLayered[1], prop.maxSurface2DLayered[2]);
		fprintf(stdout, "Maximum 3D surface dimensions:"
			"                                                      (%d, %d, %d)\n",
			prop.maxSurface3D[0], prop.maxSurface3D[1], prop.maxSurface3D[2]);
		fprintf(stdout, "Maximum Cubemap surface dimensions:"
			"                                                 %d\n", prop.maxSurfaceCubemap);
		fprintf(stdout, "Maximum Cubemap layered surface dimensions:"
			"                                         (%d, %d)\n",
			prop.maxSurfaceCubemapLayered[0], prop.maxSurfaceCubemapLayered[1]);
		fprintf(stdout, "Maximum 1D texture size: "
			"                                                           %d\n", prop.maxTexture1D);
		fprintf(stdout, "Maximum 1D layered texture dimensions:"
			"                                              (%d, %d)\n",
			prop.maxTexture1DLayered[0], prop.maxTexture1DLayered[1]);
		fprintf(stdout, "Maximum size for 1D textures bound to linear memory:"
			"                                %d\n", prop.maxTexture1DLinear);
		fprintf(stdout, "Maximum 1D mipmapped texture size:"
			"                                                  %d\n", prop.maxTexture1DMipmap);
		fprintf(stdout, "Maximum 2D texture dimensions:"
			"                                                      (%d, %d)\n",
			prop.maxTexture2D[0], prop.maxTexture2D[1]);
		fprintf(stdout, "Maximum 2D texture dimensions if texture gather operations have to be performed:"
			"    (%d, %d)\n", prop.maxTexture2DGather[0], prop.maxTexture2DGather[1]);
		fprintf(stdout, "Maximum 2D layered texture dimensions:"
			"                                              (%d, %d, %d)\n",
			prop.maxTexture2DLayered[0], prop.maxTexture2DLayered[1], prop.maxTexture2DLayered[2]);
		fprintf(stdout, "Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory:"
			"  (%d, %d, %d)\n",
			prop.maxTexture2DLinear[0], prop.maxTexture2DLinear[1], prop.maxTexture2DLinear[2]);
		fprintf(stdout, "Maximum 2D mipmapped texture dimensions:"
			"                                            (%d, %d)\n",
			prop.maxTexture2DMipmap[0], prop.maxTexture2DMipmap[1]);
		fprintf(stdout, "Maximum 3D texture dimensions:"
			"                                                      (%d, %d, %d)\n",
			prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2]);
		fprintf(stdout, "Maximum alternate 3D texture dimensions:"
			"                                            (%d, %d, %d)\n",
			prop.maxTexture3DAlt[0], prop.maxTexture3DAlt[1], prop.maxTexture3DAlt[2]);
		fprintf(stdout, "Maximum Cubemap texture dimesions:"
			"                                                  %d\n", prop.maxTextureCubemap);
		fprintf(stdout, "Maximum Cubemap layered texture dimensions:"
			"                                         (%d, %d)\n",
			prop.maxTextureCubemapLayered[0], prop.maxTextureCubemapLayered[1]);
		fprintf(stdout, "Maximum size of each dimension of a block:"
			"                                          (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		fprintf(stdout, "Maximum number of threads per block:"
			"                                                %d\n", prop.maxThreadsPerBlock);
		fprintf(stdout, "Maximum resident threads per multiprocessor:"
			"                                        %d\n", prop.maxThreadsPerMultiProcessor);
		fprintf(stdout, "Maximum pitch in bytes allowed by memory copies:"
			"                                    %lu\n", prop.memPitch);
		fprintf(stdout, "Global memory bus width in bits:"
			"                                                    %d\n", prop.memoryBusWidth);
		fprintf(stdout, "Peak memory clock frequency in kilohertz:"
			"                                           %d\n", prop.memoryClockRate);
		fprintf(stdout, "Unique identifier for a group of devices on the same multi-GPU board:"
			"               %d\n", prop.multiGpuBoardGroupID);
		fprintf(stdout, "Number of multiprocessors on device:"
			"                                                %d\n", prop.multiProcessorCount);
		fprintf(stdout, "PCI bus ID of the device:"
			"                                                           %d\n", prop.pciBusID);
		fprintf(stdout, "PCI device ID of the device:"
			"                                                        %d\n", prop.pciDeviceID);
		fprintf(stdout, "PCI domain ID of the device:"
			"                                                        %d\n", prop.pciDomainID);
		fprintf(stdout, "32-bit registers available per block:"
			"                                               %d\n", prop.regsPerBlock);
		fprintf(stdout, "32-bit registers available per multiprocessor:"
			"                                      %d\n", prop.regsPerMultiprocessor);
		fprintf(stdout, "Shared memory available per block in bytes:"
			"                                         %lu\n", prop.sharedMemPerBlock);
		fprintf(stdout, "Shared memory available per multiprocessor in bytes:"
			"                                %lu\n", prop.sharedMemPerMultiprocessor);
		fprintf(stdout, "Device supports stream priorities:    "
			"                                              %d\n", prop.streamPrioritiesSupported);
		fprintf(stdout, "Alignment requirements for surfaces:"
			"                                                %lu\n", prop.surfaceAlignment);
		fprintf(stdout, "1 if device is a Tesla device using TCC driver, 0 othrewise:"
			"                        %d\n", prop.tccDriver);
		fprintf(stdout, "Alignment requirement for textures:"
			"                                                 %lu\n", prop.textureAlignment);
		fprintf(stdout, "Pitch alignment requirement for texture references bound to pitched memory:"
			"         %lu\n", prop.texturePitchAlignment);
		fprintf(stdout, "Constant memory available on device in bytes:"
			"                                       %lu\n", prop.totalConstMem);
		fprintf(stdout, "Global memory available on device in bytes:"
			"                                         %lu\n", prop.totalGlobalMem);
		fprintf(stdout, "Device shares a unified address space with the host:"
			"                                %d\n", prop.unifiedAddressing);
		fprintf(stdout, "Warp size in threads:"
			"                                                               %d\n", prop.warpSize);
	}
	hipDeviceReset();
	return 0;
}

