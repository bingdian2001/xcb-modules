#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2017, Dalian Futures Information Technology Co., Ltd.
 *
 * Bo Wang
 * Xiaoye Meng <mengxiaoye at dce dot com dot cn>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
	int count = 0, dev;

	hipGetDeviceCount(&count);
	if (count == 0)
		printf("There are no available device(s) that support CUDA\n");
	else
		printf("Detected %d CUDA Capable device(s)\n", count);
	for (dev = 0; dev < count; ++dev) {
		hipDeviceProp_t prop;
		int dver = 0, rver = 0;

		hipSetDevice(dev);
		hipGetDeviceProperties(&prop, dev);
		printf("\nDevice %d: \"%s\"\n", dev, prop.name);
		hipDriverGetVersion(&dver);
		hipRuntimeGetVersion(&rver);
		printf("CUDA Driver Version / Runtime Version"
			"                                               %d.%d / %d.%d\n",
			dver / 1000, dver % 100 / 10, rver / 1000, rver % 100 / 10);
		printf("CUDA Major / Minor compute capability:"
			"                                              %d.%d\n", prop.major, prop.minor);
		printf("Device has ECC support enabled:"
			"                                                     %d\n", prop.ECCEnabled);
		printf("Number of asynchronous engines:"
			"                                                     %d\n", prop.asyncEngineCount);
		printf("Device can map host memory with hipHostAlloc/hipHostGetDevicePointer:"
			"             %d\n", prop.canMapHostMemory);
		printf("Clock frequency in kilohertz:"
			"                                                       %d\n", prop.clockRate);
		printf("Compute mode:           "
			"                                                            %d\n", prop.computeMode);
		printf("Device can possibly execute multiple kernels concurrently:"
			"                          %d\n", prop.concurrentKernels);
		printf("Device supports caching globals in L1:"
			"                                              %d\n", prop.globalL1CacheSupported);
		printf("Device is integrated as opposed to discrete:"
			"                                        %d\n", prop.integrated);
		printf("Device is on a multi-GPU board:"
			"                                                     %d\n", prop.isMultiGpuBoard);
		printf("Specified whether there is a run time limit on kernels:"
			"                             %d\n", prop.kernelExecTimeoutEnabled);
		printf("Size of L2 cache in bytes:"
			"                                                          %d\n", prop.l2CacheSize);
		printf("Device supports caching locals in L1:"
			"                                               %d\n", prop.localL1CacheSupported);
		printf("Device supports allocating managed memory on this system:"
			"                           %d\n", prop.managedMemory);
		printf("Maximum size of each dimension of a grid:"
			"                                           (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Maximum 1D surface size: "
			"                                                           %d\n", prop.maxSurface1D);
		printf("Maximum 1D layered surface dimensions:"
			"                                              (%d, %d)\n",
			prop.maxSurface1DLayered[0], prop.maxSurface1DLayered[1]);
		printf("Maximum 2D surface dimensions:"
			"                                                      (%d, %d)\n",
			prop.maxSurface2D[0], prop.maxSurface2D[1]);
		printf("Maximum 2D layered surface dimensions:"
			"                                              (%d, %d, %d)\n",
			prop.maxSurface2DLayered[0], prop.maxSurface2DLayered[1], prop.maxSurface2DLayered[2]);
		printf("Maximum 3D surface dimensions:"
			"                                                      (%d, %d, %d)\n",
			prop.maxSurface3D[0], prop.maxSurface3D[1], prop.maxSurface3D[2]);
		printf("Maximum Cubemap surface dimensions:"
			"                                                 %d\n", prop.maxSurfaceCubemap);
		printf("Maximum Cubemap layered surface dimensions:"
			"                                         (%d, %d)\n",
			prop.maxSurfaceCubemapLayered[0], prop.maxSurfaceCubemapLayered[1]);
		printf("Maximum 1D texture size: "
			"                                                           %d\n", prop.maxTexture1D);
		printf("Maximum 1D layered texture dimensions:"
			"                                              (%d, %d)\n",
			prop.maxTexture1DLayered[0], prop.maxTexture1DLayered[1]);
		printf("Maximum size for 1D textures bound to linear memory:"
			"                                %d\n", prop.maxTexture1DLinear);
		printf("Maximum 1D mipmapped texture size:"
			"                                                  %d\n", prop.maxTexture1DMipmap);
		printf("Maximum 2D texture dimensions:"
			"                                                      (%d, %d)\n",
			prop.maxTexture2D[0], prop.maxTexture2D[1]);
		printf("Maximum 2D texture dimensions if texture gather operations have to be performed:"
			"    (%d, %d)\n", prop.maxTexture2DGather[0], prop.maxTexture2DGather[1]);
		printf("Maximum 2D layered texture dimensions:"
			"                                              (%d, %d, %d)\n",
			prop.maxTexture2DLayered[0], prop.maxTexture2DLayered[1], prop.maxTexture2DLayered[2]);
		printf("Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory:"
			"  (%d, %d, %d)\n",
			prop.maxTexture2DLinear[0], prop.maxTexture2DLinear[1], prop.maxTexture2DLinear[2]);
		printf("Maximum 2D mipmapped texture dimensions:"
			"                                            (%d, %d)\n",
			prop.maxTexture2DMipmap[0], prop.maxTexture2DMipmap[1]);
		printf("Maximum 3D texture dimensions:"
			"                                                      (%d, %d, %d)\n",
			prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2]);
		printf("Maximum alternate 3D texture dimensions:"
			"                                            (%d, %d, %d)\n",
			prop.maxTexture3DAlt[0], prop.maxTexture3DAlt[1], prop.maxTexture3DAlt[2]);
		printf("Maximum Cubemap texture dimesions:"
			"                                                  %d\n", prop.maxTextureCubemap);
		printf("Maximum Cubemap layered texture dimensions:"
			"                                         (%d, %d)\n",
			prop.maxTextureCubemapLayered[0], prop.maxTextureCubemapLayered[1]);
		printf("Maximum size of each dimension of a block:"
			"                                          (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Maximum number of threads per block:"
			"                                                %d\n", prop.maxThreadsPerBlock);
		printf("Maximum resident threads per multiprocessor:"
			"                                        %d\n", prop.maxThreadsPerMultiProcessor);
		printf("Maximum pitch in bytes allowed by memory copies:"
			"                                    %lu\n", prop.memPitch);
		printf("Global memory bus width in bits:"
			"                                                    %d\n", prop.memoryBusWidth);
		printf("Peak memory clock frequency in kilohertz:"
			"                                           %d\n", prop.memoryClockRate);
		printf("Unique identifier for a group of devices on the same multi-GPU board:"
			"               %d\n", prop.multiGpuBoardGroupID);
		printf("Number of multiprocessors on device:"
			"                                                %d\n", prop.multiProcessorCount);
		printf("PCI bus ID of the device:"
			"                                                           %d\n", prop.pciBusID);
		printf("PCI device ID of the device:"
			"                                                        %d\n", prop.pciDeviceID);
		printf("PCI domain ID of the device:"
			"                                                        %d\n", prop.pciDomainID);
		printf("32-bit registers available per block:"
			"                                               %d\n", prop.regsPerBlock);
		printf("32-bit registers available per multiprocessor:"
			"                                      %d\n", prop.regsPerMultiprocessor);
		printf("Shared memory available per block in bytes:"
			"                                         %lu\n", prop.sharedMemPerBlock);
		printf("Shared memory available per multiprocessor in bytes:"
			"                                %lu\n", prop.sharedMemPerMultiprocessor);
		printf("Device supports stream priorities:    "
			"                                              %d\n", prop.streamPrioritiesSupported);
		printf("Alignment requirements for surfaces:"
			"                                                %lu\n", prop.surfaceAlignment);
		printf("1 if device is a Tesla device using TCC driver, 0 othrewise:"
			"                        %d\n", prop.tccDriver);
		printf("Alignment requirement for textures:"
			"                                                 %lu\n", prop.textureAlignment);
		printf("Pitch alignment requirement for texture references bound to pitched memory:"
			"         %lu\n", prop.texturePitchAlignment);
		printf("Constant memory available on device in bytes:"
			"                                       %lu\n", prop.totalConstMem);
		printf("Global memory available on device in bytes:"
			"                                         %lu\n", prop.totalGlobalMem);
		printf("Device shares a unified address space with the host:"
			"                                %d\n", prop.unifiedAddressing);
		printf("Warp size in threads:"
			"                                                               %d\n", prop.warpSize);
	}
	hipDeviceReset();
	return 0;
}

