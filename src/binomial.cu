#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016, Dalian Futures Information Technology Co., Ltd.
 *
 * Bo Wang
 * Xiaoye Meng <mengxiaoye at dce dot com dot cn>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <hip/hip_runtime.h>
#include "impv.h"
#include "brent.h"

/* FIXME */
#define MAX_STEPS   4096
#define CACHE_SIZE  256
#define TIME_STEPS  16
#define CACHE_DELTA (2 * TIME_STEPS)
#define CACHE_STEP  (CACHE_SIZE - CACHE_DELTA)

/* FIXME */
static __device__ double d_pricebuf[MAX_STEPS + 16];
static __device__ double  d_callbuf[MAX_STEPS + 16];
static __device__ double   d_putbuf[MAX_STEPS + 16];

/* FIXME */
static __global__ void bi_amer_call(double spot, double strike, double vdt, double pu, double pd,
	int steps, double *d_callval) {
	__shared__ double pricea[CACHE_SIZE + 1];
	__shared__ double priceb[CACHE_SIZE + 1];
	__shared__ double  calla[CACHE_SIZE + 1];
	__shared__ double  callb[CACHE_SIZE + 1];
	const int tid = threadIdx.x;
	int i, base;

	for (i = tid; i <= steps; i += CACHE_SIZE) {
		d_pricebuf[i] = spot * exp(vdt * (2.0 * i - steps));
		d_callbuf[i]  = spot * exp(vdt * (2.0 * i - steps)) - strike;
		d_callbuf[i]  = d_callbuf[i] > 0.0 ? d_callbuf[i] : 0.0;
	}
	for (i = steps; i > 0; i -= CACHE_DELTA)
		for (base = 0; base < i; base += CACHE_STEP) {
			int start = min(CACHE_SIZE - 1, i - base);
			int end   = start - CACHE_DELTA;
			int k;

			__syncthreads();
			if (tid <= start) {
				pricea[tid] = d_pricebuf[base + tid];
				calla[tid]  = d_callbuf[base + tid];
			}
			for (k = start - 1; k >= end;) {
				double callval;

				__syncthreads();
				priceb[tid] = exp(-vdt) * pricea[tid + 1];
				callval = pu * calla[tid + 1] + pd * calla[tid];
				callb[tid] = callval > (priceb[tid] - strike)
					? callval : (priceb[tid] - strike);
				k--;
				__syncthreads();
				pricea[tid] = exp(-vdt) * priceb[tid + 1];
				callval = pu * callb[tid + 1] + pd * callb[tid];
				calla[tid] = callval > (pricea[tid] - strike)
					? callval : (pricea[tid] - strike);
				k--;
			}
			__syncthreads();
			if (tid <= end) {
				d_pricebuf[base + tid] = pricea[tid];
				d_callbuf[base + tid]  = calla[tid];
			}
		}
	if (threadIdx.x == 0)
		*d_callval = calla[0];
}

/* FIXME */
static __global__ void bi_amer_put(double spot, double strike, double vdt, double pu, double pd,
	int steps, double *d_putval) {
	__shared__ double pricea[CACHE_SIZE + 1];
	__shared__ double priceb[CACHE_SIZE + 1];
	__shared__ double   puta[CACHE_SIZE + 1];
	__shared__ double   putb[CACHE_SIZE + 1];
	const int tid = threadIdx.x;
	int i, base;

	for (i = tid; i <= steps; i += CACHE_SIZE) {
		d_pricebuf[i] = spot * exp(vdt * (2.0 * i - steps));
		d_putbuf[i]   = strike - spot * exp(vdt * (2.0 * i - steps));
		d_putbuf[i]   = d_putbuf[i] > 0.0 ? d_putbuf[i] : 0.0;
	}
	for (i = steps; i > 0; i -= CACHE_DELTA)
		for (base = 0; base < i; base += CACHE_STEP) {
			int start = min(CACHE_SIZE - 1, i - base);
			int end   = start - CACHE_DELTA;
			int k;

			__syncthreads();
			if (tid <= start) {
				pricea[tid] = d_pricebuf[base + tid];
				puta[tid]   = d_putbuf[base + tid];
			}
			for (k = start - 1; k >= end;) {
				double putval;

				__syncthreads();
				priceb[tid] = exp(-vdt) * pricea[tid + 1];
				putval = pu * puta[tid + 1] + pd * puta[tid];
				putb[tid] = putval > (strike - priceb[tid])
					? putval : (strike - priceb[tid]);
				k--;
				__syncthreads();
				pricea[tid] = exp(-vdt) * priceb[tid + 1];
				putval = pu * putb[tid + 1] + pd * putb[tid];
				puta[tid] = putval > (strike - pricea[tid])
					? putval : (strike - pricea[tid]);
				k--;
			}
			__syncthreads();
			if (tid <= end) {
				d_pricebuf[base + tid] = pricea[tid];
				d_putbuf[base + tid]   = puta[tid];
			}
		}
	if (threadIdx.x == 0)
		*d_putval = puta[0];
}

/* FIXME */
double bi_cuda_amer_call(double spot, double strike, double r, double d, double vol, double expiry, int steps) {
	double dt = expiry / steps;
	/* interest rate for each step */
	double R = exp(r * dt);
	/* inverse of interest rate */
	double Rinv = 1.0 / R;
	double vdt = vol * sqrt(dt);
	/* up movement */
	double up = exp(vdt);
	/* down movement */
	double dn = 1.0 / up;
	double p_up = (exp((r - d) * dt) - dn) / (up - dn);
	double p_dn = 1.0 - p_up;
	double *d_callval, res = 0.0;

	hipMalloc((void **)&d_callval, sizeof (double));
	bi_amer_call<<<1, CACHE_SIZE>>>(spot, strike, vdt, Rinv * p_up, Rinv * p_dn, steps, d_callval);
	hipMemcpy(&res, d_callval, sizeof (double), hipMemcpyDeviceToHost);
	hipFree(d_callval);
	return res;
}

/* FIXME */
double bi_cuda_amer_put(double spot, double strike, double r, double d, double vol, double expiry, int steps) {
	double dt = expiry / steps;
	/* interest rate for each step */
	double R = exp(r * dt);
	/* inverse of interest rate */
	double Rinv = 1.0 / R;
	double vdt = vol * sqrt(dt);
	/* up movement */
	double up = exp(vdt);
	/* down movement */
	double dn = 1.0 / up;
	double p_up = (exp((r - d) * dt) - dn) / (up - dn);
	double p_dn = 1.0 - p_up;
	double *d_putval, res = 0.0;

	hipMalloc((void **)&d_putval, sizeof (double));
	bi_amer_put<<<1, CACHE_SIZE>>>(spot, strike, vdt, Rinv * p_up, Rinv * p_dn, steps, d_putval);
	hipMemcpy(&res, d_putval, sizeof (double), hipMemcpyDeviceToHost);
	hipFree(d_putval);
	return res;
}

/* FIXME */
double impv_bi_cuda(double spot, double strike, double r, double d, double expiry, int steps,
	double price, int type) {
	double low = 0.000001, high = 0.3, ce;

	/* FIXME */
	if (type != AMER_CALL && type != AMER_PUT)
		return NAN;
	ce = type == AMER_CALL ? bi_cuda_amer_call(spot, strike, r, d, high, expiry, steps) :
		bi_cuda_amer_put(spot, strike, r, d, high, expiry, steps);
	while (ce < price) {
		high *= 2.0;
		if (high > 1e10)
			return NAN;
		ce = type == AMER_CALL ? bi_cuda_amer_call(spot, strike, r, d, high, expiry, steps) :
			bi_cuda_amer_put(spot, strike, r, d, high, expiry, steps);
	}
	return type == AMER_CALL
		? brent(low, high, price, NULL, bi_cuda_amer_call, NULL, spot, strike, r, d, expiry, 0, steps)
		: brent(low, high, price, NULL, bi_cuda_amer_put,  NULL, spot, strike, r, d, expiry, 0, steps);
}

