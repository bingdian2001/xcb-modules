
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2013-2016, Dalian Futures Information Technology Co., Ltd.
 *
 * Bo Wang
 * Xiaoye Meng <mengxiaoye at dce dot com dot cn>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

/* FIXME */
#define MAX_STEPS   4096
#define CACHE_SIZE  256
#define TIME_STEPS  16
#define CACHE_DELTA (2 * TIME_STEPS)
#define CACHE_STEP  (CACHE_SIZE - CACHE_DELTA)

/* FIXME */
static __device__ double d_pricebuf[MAX_STEPS + 16];
static __device__ double  d_callbuf[MAX_STEPS + 16];
static __device__ double d_callval;

/* FIXME */
static __global__ void bi_amer_call(double spot, double strike, double vdt, double pu, double pd, int steps) {
	__shared__ double pricea[CACHE_SIZE + 1];
	__shared__ double priceb[CACHE_SIZE + 1];
	__shared__ double  calla[CACHE_SIZE + 1];
	__shared__ double  callb[CACHE_SIZE + 1];
	const int tid = threadIdx.x;
	int i, base;

	for (i = tid; i <= steps; i += CACHE_SIZE) {
		d_pricebuf[i] = spot * exp(vdt * (2.0 * i - steps));
		d_callbuf[i]  = spot * exp(vdt * (2.0 * i - steps)) - strike;
		d_callbuf[i]  = d_callbuf[i] > 0.0 ? d_callbuf[i] : 0.0;
	}
	for (i = steps; i > 0; i -= CACHE_DELTA)
		for (base = 0; base < i; base += CACHE_STEP) {
			int start = min(CACHE_SIZE - 1, i - base);
			int end   = start - CACHE_DELTA;
			int k;

			__syncthreads();
			if (tid <= start) {
				pricea[tid] = d_pricebuf[base + tid];
				calla[tid]  = d_callbuf[base + tid];
			}
			for (k = start - 1; k >= end;) {
				double callval;

				__syncthreads();
				priceb[tid] = exp(-vdt) * pricea[tid + 1];
				callval = pu * calla[tid + 1] + pd * calla[tid];
				callb[tid] = callval > (priceb[tid] - strike)
					? callval : (priceb[tid] - strike);
				k--;
				__syncthreads();
				pricea[tid] = exp(-vdt) * priceb[tid + 1];
				callval = pu * callb[tid + 1] + pd * callb[tid];
				calla[tid] = callval > (pricea[tid] - strike)
					? callval : (pricea[tid] - strike);
				k--;
			}
			__syncthreads();
			if (tid <= end) {
				d_pricebuf[base + tid] = pricea[tid];
				d_callbuf[base + tid]  = calla[tid];
			}
		}
	if (threadIdx.x == 0) {
		d_callval = calla[0];
	}
}

/* FIXME */
void bi_cuda_amer_call(double spot, double strike, double r, double d, double vol, double expiry,
	int steps, double *res) {
	double dt = expiry / steps;
	/* interest rate for each step */
	double R = exp(r * dt);
	/* inverse of interest rate */
	double Rinv = 1.0 / R;
	double vdt = vol * sqrt(dt);
	/* up movement */
	double up = exp(vdt);
	/* down movement */
	double dn = 1.0 / up;
	double p_up = (exp((r - d) * dt) - dn) / (up - dn);
	double p_dn = 1.0 - p_up;
	double pu = Rinv * p_up;
	double pd = Rinv * p_dn;

	bi_amer_call<<<1, CACHE_SIZE>>>(spot, strike, vdt, pu, pd, steps);
	hipMemcpy(res, &d_callval, 1, hipMemcpyDeviceToHost);
}

